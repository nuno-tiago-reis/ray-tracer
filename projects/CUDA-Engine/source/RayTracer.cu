#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

// CUDA definitions
#include <hip/hip_runtime.h>
// CUB definitions
#include <cub.cuh>

// Math Includes 
#include <hip/hip_vector_types.h>
#include <math_functions.h>
// Vector Includes
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

// C++ Includes
#include <stdio.h>
// Utility Includes
#include "Utility.h"
#include "Constants.h"

// Secondary Ray Depth
static const int depth = 0;
// Air Refraction Index
static const float refractionIndex = 1.0f;

// Ray testing Constant
//static const float epsilon = 0.01f;

// Temporary Storage
static void *scanTemporaryStorage = NULL;
static size_t scanTemporaryStoreBytes = 0;

static void *radixSortTemporaryStorage = NULL;
static size_t radixSortTemporaryStoreBytes = 0;

// Ray indexing Constants
__constant__ __device__ static const float bit_mask_1_4_f = 15.0f;
//__constant__ __device__ static const float bit_mask_1_5_f = 31.0f;

__constant__ __device__ static const float half_bit_mask_1_4_f = 7.0f;
//__constant__ __device__ static const float half_bit_mask_1_5_f = 15.0f;

// OpenGL Diffuse and Specular Textures
texture<float4, hipTextureType2D, hipReadModeElementType> diffuseTexture;
texture<float4, hipTextureType2D, hipReadModeElementType> specularTexture;
// OpenGL Fragment Position and Normal Textures
texture<float4, hipTextureType2D, hipReadModeElementType> fragmentPositionTexture;
texture<float4, hipTextureType2D, hipReadModeElementType> fragmentNormalTexture;

// CUDA Triangle Textures
texture<float4, 1, hipReadModeElementType> trianglePositionsTexture;
texture<float4, 1, hipReadModeElementType> triangleNormalsTexture;
texture<float2, 1, hipReadModeElementType> triangleTextureCoordinatesTexture;

// CUDA Triangle ID Textures
texture<int1, 1, hipReadModeElementType> triangleObjectIDsTexture;
texture<int1, 1, hipReadModeElementType> triangleMaterialIDsTexture;

// CUDA Material Textures
texture<float4, 1, hipReadModeElementType> materialDiffusePropertiesTexture;
texture<float4, 1, hipReadModeElementType> materialSpecularPropertiesTexture;

// CUDA Light Textures
texture<float4, 1, hipReadModeElementType> lightPositionsTexture;
texture<float4, 1, hipReadModeElementType> lightColorsTexture;
texture<float2, 1, hipReadModeElementType> lightIntensitiesTexture;

// Ray structure
struct Ray {

	float3 origin;
	float3 direction;
	float3 inverseDirection;

	__device__ Ray() {};
	__device__ Ray(const float3 &o,const float3 &d) {

		origin = o;
		direction = d;
		direction = normalize(direction);
		inverseDirection = make_float3(1.0/direction.x, 1.0/direction.y, 1.0/direction.z);
	}
};

struct HitRecord {

	float time;

	float3 color;

	float3 point;
	float3 normal;

	int triangleIndex;

	__device__ HitRecord(const float3 &c) {

			time = UINT_MAX;

			color = c;

			point = make_float3(0,0,0);
			normal = make_float3(0,0,0);

			triangleIndex = -1; 
	}

	__device__ void resetTime() {
		
			time = UINT_MAX;

			point = make_float3(0,0,0);
			normal = make_float3(0,0,0);

			triangleIndex = -1;
	}
};

// Converts 8-bit integer to floating point rgb color
__device__ float3 intToRgb(int color) {

	float red	= color & 255;
	float green	= (color >> 8) & 255;
	float blue	= (color >> 16) & 255;

	return make_float3(red, green, blue);
}

// Converts floating point rgb color to 8-bit integer
__device__ int rgbToInt(float red, float green, float blue) {

	red		= clamp(red,	0.0f, 255.0f);
	green	= clamp(green,	0.0f, 255.0f);
	blue	= clamp(blue,	0.0f, 255.0f);

	return (int(red)) | (int(green)<<8) | (int(blue)<<16); // notice switch red and blue to counter the GL_BGRA
}

// Converts a Direction Vector to Spherical Coordinates
__device__ float2 vectorToSpherical(float3 direction) {

	float azimuth = atan(direction.y / direction.x) * 2.0f;
	float polar = acos(direction.z);

	return make_float2(azimuth,polar);
}

// Converts Spherical Coordinates to a Direction Vector
__device__ float3 sphericalToVector(float2 spherical) {

	float x = cos(spherical.x) * sin(spherical.y);
	float y = sin(spherical.x) * sin(spherical.y);
	float z = cos(spherical.y);

	return make_float3(x,y,z);
}

// Converts a ray to an integer hash value
__device__ int rayToIndex(float3 origin, float3 direction) {

	int index = 0;

	// Convert the Direction to Spherical Coordinates
	index = (unsigned int)clamp((atan(direction.y / direction.x) + HALF_PI) * RADIANS_TO_DEGREES * 2.0f, 0.0f, 360.0f);
	index = (index << 9) | (unsigned int)clamp(acos(direction.z) * RADIANS_TO_DEGREES, 0.0f, 180.0f);

	// Clamp the Origin to the 0-15 range
	index = (index << 4) | (unsigned int)clamp(origin.x + half_bit_mask_1_4_f, 0.0f, bit_mask_1_4_f);
	index = (index << 4) | (unsigned int)clamp(origin.y + half_bit_mask_1_4_f, 0.0f, bit_mask_1_4_f);
	index = (index << 4) | (unsigned int)clamp(origin.z + half_bit_mask_1_4_f, 0.0f, bit_mask_1_4_f);
	//index = (index << 5) | (unsigned int)clamp(origin.y + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);
	//index = (index << 5) | (unsigned int)clamp(origin.z + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);

	index++;

	return index;
}

// Ray - BoundingBox Intersection Code
__device__ int RayBoxIntersection(const float3 &BBMin, const float3 &BBMax, const float3 &RayOrigin, const float3 &RayDirectionInverse, float &tmin, float &tmax) {

	float l1   = (BBMin.x - RayOrigin.x) * RayDirectionInverse.x;
	float l2   = (BBMax.x - RayOrigin.x) * RayDirectionInverse.x;
	tmin = fminf(l1,l2);
	tmax = fmaxf(l1,l2);

	l1   = (BBMin.y - RayOrigin.y) * RayDirectionInverse.y;
	l2   = (BBMax.y - RayOrigin.y) * RayDirectionInverse.y;
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);

	l1   = (BBMin.z - RayOrigin.z) * RayDirectionInverse.z;
	l2   = (BBMax.z - RayOrigin.z) * RayDirectionInverse.z;
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);

	return ((tmax >= tmin) && (tmax >= 0.0f));
}

// Ray - Triangle Intersection Code
__device__ float RayTriangleIntersection(const Ray &ray, const float3 &vertex0, const float3 &edge1, const float3 &edge2) {  

	float3 tvec = ray.origin - vertex0;  
	float3 pvec = cross(ray.direction, edge2);  

	float  determinant  = dot(edge1, pvec);  
	determinant = __fdividef(1.0f, determinant);  

	// First Test
	float u = dot(tvec, pvec) * determinant;  
	if (u < 0.0f || u > 1.0f)  
		return -1.0f;  

	// Second Test
	float3 qvec = cross(tvec, edge1);  

	float v = dot(ray.direction, qvec) * determinant;  
	if (v < 0.0f || (u + v) > 1.0f)  
		return -1.0f;  

	return dot(edge2, qvec) * determinant;  
}  

// Implementation of the Matrix Multiplication
__global__ void UpdateVertex(
							// Updated Normal Matrices Array
							float* modelMatricesArray,
							// Updated Normal Matrices Array
							float* normalMatricesArray,
							// Updated Triangle Positions Array
							float4* trianglePositionsArray,
							// Updated Triangle Normals Array
							float4* triangleNormalsArray,
							// Total Number of Vertices in the Scene
							int vertexTotal) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= vertexTotal)
		return;

	// Matrices ID
	int matrixID = tex1Dfetch(triangleObjectIDsTexture, x).x;

	// Model Matrix - Multiply each Vertex Position by it.
	float modelMatrix[16];

	for(int i=0; i<16; i++)
		modelMatrix[i] = modelMatricesArray[matrixID * 16 + i];
	
	float4 vertex = tex1Dfetch(trianglePositionsTexture, x);

	float updatedVertex[4];

	for(int i=0; i<4; i++) {

		updatedVertex[i] = 0.0f;
		updatedVertex[i] += modelMatrix[i * 4 + 0] * vertex.x;
		updatedVertex[i] += modelMatrix[i * 4 + 1] * vertex.y;
		updatedVertex[i] += modelMatrix[i * 4 + 2] * vertex.z;
		updatedVertex[i] += modelMatrix[i * 4 + 3] * vertex.w;
	}
	
	// Store the updated Vertex Position.
	trianglePositionsArray[x] = make_float4(updatedVertex[0], updatedVertex[1], updatedVertex[2], matrixID);

	// Normal Matrix - Multiply each Vertex Normal by it.
	float normalMatrix[16];

	for(int i=0; i<16; i++)
		normalMatrix[i] = normalMatricesArray[matrixID * 16 + i];

	float4 normal = tex1Dfetch(triangleNormalsTexture, x);

	float updatedNormal[4];

	for(int i=0; i<4; i++) {

		updatedNormal[i] = 0.0f;
		updatedNormal[i] += normalMatrix[i * 4 + 0] * normal.x;
		updatedNormal[i] += normalMatrix[i * 4 + 1] * normal.y;
		updatedNormal[i] += normalMatrix[i * 4 + 2] * normal.z;
		updatedNormal[i] += normalMatrix[i * 4 + 3] * normal.w;
	}

	// Store the updated Vertex Normal.
	triangleNormalsArray[x] = make_float4(normalize(make_float3(updatedNormal[0], updatedNormal[1], updatedNormal[2])), 0.0f);
}

//	Ray index Array	
__global__ void CreateRays(// Input Array containing the unsorted Rays
							float3* rayArray,
							// Screen Dimensions
							int windowWidth, int windowHeight,
							// Total number of Light Sources in the Scene
							int lightTotal,
							// Cameras Position in the Scene
							float3 cameraPosition,
							// Output Array containing the exclusing scan result
							int* headFlagsArray, 
							// Output Arrays containing the Ray Indices [Keys = Hashes, Values = Indices]
							int* rayIndexKeysArray, 
							int* rayIndexValuesArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= windowWidth || y >= windowHeight)
		return;

	int rayBase = windowWidth * windowHeight;
	int rayOffset = x + y * windowWidth;

	// Fragment Position and Normal - Sent from the OpenGL Rasterizer
	float3 fragmentPosition = make_float3(tex2D(fragmentPositionTexture, x,y));
	float3 fragmentNormal = normalize(make_float3(tex2D(fragmentNormalTexture, x,y)));

	if(length(fragmentPosition) != 0.0f) {
		
		// Ray Direction Creation
		float3 rayReflectionDirection = reflect(normalize(fragmentPosition-cameraPosition), normalize(fragmentNormal));
		float3 rayRefractionDirection = refract(normalize(fragmentPosition-cameraPosition), normalize(fragmentNormal), 1.0f / 1.52f);
		
		// Light Positions - Sent from the CPU
		float3 shadowRayPositions[LIGHT_SOURCE_MAXIMUM];
		float3 shadowRayDirections[LIGHT_SOURCE_MAXIMUM];

		// Create the Reflection and Refraction Rays and store their directions
		rayArray[(rayOffset * 2)] = fragmentPosition;
		rayArray[(rayOffset * 2) + 1] = rayReflectionDirection;

		rayArray[(rayBase + rayOffset) * 2] = fragmentPosition;
		rayArray[(rayBase + rayOffset) * 2 + 1] = rayRefractionDirection;

		// Create the Shadow Rays
		for(int l = 0; l < lightTotal; l++) {

			// Calculate the Shadow Rays Position and Direction
			shadowRayPositions[l] = make_float3(tex1Dfetch(lightPositionsTexture, l));
			shadowRayDirections[l] = normalize(shadowRayPositions[l] - fragmentPosition);

			// Diffuse Factor
			float diffuseFactor = max(dot(shadowRayDirections[l], fragmentNormal), 0.0f);
			clamp(diffuseFactor, 0.0f, 1.0f);
				
			// Store the Shadow Rays its direction
			if(diffuseFactor <= 0.0f)
				shadowRayDirections[l] = make_float3(0.0f);
			
			rayArray[(rayBase * (2 + l) + rayOffset) * 2] = fragmentPosition;
			rayArray[(rayBase * (2 + l) + rayOffset) * 2 + 1] = shadowRayDirections[l];
		}

		// Store the Reflection and Refraction Ray indices
		rayIndexKeysArray[rayOffset] = rayToIndex(fragmentPosition, rayReflectionDirection);
		rayIndexValuesArray[rayOffset] = rayOffset;

		rayIndexKeysArray[rayBase + rayOffset] = rayToIndex(fragmentPosition, rayRefractionDirection);
		rayIndexValuesArray[rayBase + rayOffset] = rayBase + rayOffset;

		// Store the Shadow Ray Indices
		for(int l = 0; l < lightTotal; l++) {
				
			// Create the Shadow Ray and store its direction
			if(length(shadowRayDirections[l]) > 0.0f) {

				rayIndexKeysArray[rayBase * (2 + l) + rayOffset] = rayToIndex(shadowRayPositions[l], shadowRayDirections[l]);
				rayIndexValuesArray[rayBase * (2 + l) + rayOffset] = rayBase * (2 + l) + rayOffset;
			}
			else {

				rayIndexKeysArray[rayBase * (2 + l) + rayOffset] = 0;
				rayIndexValuesArray[rayBase * (2 + l) + rayOffset] = 0;
			}
		}
		
		// Clean the Shadow Ray Index storage
		for(int l = lightTotal; l < LIGHT_SOURCE_MAXIMUM; l++) {
		
			rayIndexKeysArray[rayBase * (2 + l) + rayOffset] = 0;
			rayIndexValuesArray[rayBase * (2 + l) + rayOffset] = 0;
		}

		// Store the Reflection and Refraction Ray flags
		headFlagsArray[rayOffset] = 0;
		headFlagsArray[rayBase + rayOffset] = 0;

		// Store the Shadow Ray Indices
		for(int l = 0; l < lightTotal; l++) {
			
			// Create the Shadow Ray and store its direction
			if(length(shadowRayDirections[l]) > 0.0f)
				headFlagsArray[rayBase * (2 + l) + rayOffset] = 0;
			else
				headFlagsArray[rayBase * (2 + l) + rayOffset] = 1;
		}
		
		// Clean the Shadow Ray Index storage
		for(int l = lightTotal; l < LIGHT_SOURCE_MAXIMUM; l++) 	
			headFlagsArray[rayBase * (2 + l) + rayOffset] = 1;
	}
	else {		

		// Clear the Reflection and Refraction Ray Indices
		rayIndexKeysArray[rayOffset] = 0;
		rayIndexValuesArray[rayOffset] = 0;
		
		rayIndexKeysArray[rayBase + rayOffset] = 0;
		rayIndexValuesArray[rayBase + rayOffset] = 0;

		// Clean the Shadow Ray Indices		
		for(int l = 0; l < LIGHT_SOURCE_MAXIMUM; l++)  {
		
			rayIndexKeysArray[rayBase * (2 + l) + rayOffset] = 0;
			rayIndexValuesArray[rayBase * (2 + l) + rayOffset] = 0;
		}

		// Clear the Reflection and Refraction Ray Flags
		headFlagsArray[rayOffset] = 1;
		headFlagsArray[rayBase + rayOffset] = 1;

		// Clear the Shadow Ray Flags	
		for(int l = 0; l < LIGHT_SOURCE_MAXIMUM; l++) 
			headFlagsArray[rayBase * (2 + l) + rayOffset] = 1;
	}
}

// Implementation of the Ray Trimming
__global__ void TrimRays(	
							// Input Arrays containing the untrimmed Ray Indices [Keys = Hashes, Values = Indices]
							int* rayIndexKeysArray, 
							int* rayIndexValuesArray,
							// Total number of Rays
							int screenDimensions,
							// Auxiliary Array containing the exclusing scan result
							int* inclusiveScanArray, 
							// Output Arrays containing the trimmed Ray Indices [Keys = Hashes, Values = Indices]
							int* trimmedRayIndexKeysArray, 
							int* trimmedRayIndexValuesArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= screenDimensions)
		return;

	int startingPosition = 0;

	// Initial Position
	if(x == 0 && inclusiveScanArray[0] == 0) {

		startingPosition = 1;

		trimmedRayIndexKeysArray[0] = rayIndexKeysArray[0];
		trimmedRayIndexValuesArray[0] = rayIndexValuesArray[0];
	}

	// Remaining Positions
	for(int i=startingPosition; i<RAYS_PER_PIXEL_MAXIMUM; i++) {

		int currentPosition = x * RAYS_PER_PIXEL_MAXIMUM + i;

		// Sum Array Offsets
		int currentOffset = inclusiveScanArray[currentPosition];
		int previousOffset = inclusiveScanArray[currentPosition - 1];

		// If the Current and the Next Scan value are the same then shift the Ray
		if(currentOffset == previousOffset) {
		
			trimmedRayIndexKeysArray[currentPosition - currentOffset] = rayIndexKeysArray[currentPosition];
			trimmedRayIndexValuesArray[currentPosition - currentOffset] = rayIndexValuesArray[currentPosition];
		}
	}
}
	

// Implementation of the Ray Compression
__global__ void CreateChunkFlags(	
							// Input Arrays containing the trimmed Ray Indices [Keys = Hashes, Values = Indices]
							int* trimmedRayIndexKeysArray, 
							int* trimmedRayIndexValuesArray,
							// Total number of Rays
							int rayTotal,
							// Output Array containing the Chunk Head Flags
							int* headFlagsArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= rayTotal)
		return;

	int startingPosition = 0;

	// Initial Position
	if(x == 0) {

		startingPosition = 1;

		headFlagsArray[x] = 1;
	}

	// Remaining Positions
	for(int i=startingPosition; i<CHUNK_DIVISION; i++) {

		int currentPosition = x * CHUNK_DIVISION + i;

		if(currentPosition >= rayTotal)
			return;
	
		// Ray Hashes
		int currentHash = trimmedRayIndexKeysArray[currentPosition];
		int previousHash = trimmedRayIndexKeysArray[currentPosition - 1];

		// If the Current and Previous Ray Hashes are different, store the Head Flag
		if(currentHash != previousHash)
			headFlagsArray[currentPosition] = 1;
		else
			headFlagsArray[currentPosition] = 0;
	}
}

__global__ void CreateChunkBases(	
							// Input Arrays containing the trimmed Ray Indices [Keys = Hashes, Values = Indices]
							int* trimmedRayIndexKeysArray, 
							int* trimmedRayIndexValuesArray,
							// Total number of Rays
							int rayTotal,
							// Auxiliary Array containing the head flags result
							int* headFlagsArray, 
							// Auxiliary Array containing the exclusing scan result
							int* scanArray, 
							// Output Array containing the Ray Chunk Bases
							int* chunkBasesArray,
							// Output Arrays containing the Ray Chunks  [Keys = Hashes, Values = Indices]
							int* chunkIndexKeysArray, 
							int* chunkIndexValuesArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	
	// Remaining Positions
	for(int i=0; i<CHUNK_DIVISION; i++) {

		int currentPosition = x * CHUNK_DIVISION + i ;

		if(currentPosition >= rayTotal)
			return;
		
		// If the Head Flag isn't 1, continue;
		if(headFlagsArray[currentPosition] == 0)
			continue;

		// Store the Position of the Chunk
		int position = scanArray[currentPosition] - 1;

		// Store the Ray Base for the Chunk
		chunkBasesArray[position] = currentPosition; 
	
		// Store the Ray Hash and the Chunk Position for the Chunk
		chunkIndexKeysArray[position] = trimmedRayIndexKeysArray[currentPosition];
		chunkIndexValuesArray[position] = position;
	}
}

__global__ void CreateChunkSizes(
							// Input Array containing the Ray Chunk Bases
							int* chunkBasesArray,
							// Total number of Ray Chunks
							int chunkTotal,
							// Total number of Rays
							int rayTotal,
							// Output Array containing the Ray Chunks Sizes
							int* chunkSizesArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= chunkTotal)
		return;

	// Final Position
	if(x == chunkTotal - 1) {

		// Chunk Bases
		int currentBase = chunkBasesArray[x];
	
		chunkSizesArray[x] = rayTotal - currentBase;
	}
	else {
		
		// Chunk Bases
		int currentBase = chunkBasesArray[x];
		int nextBase = chunkBasesArray[x+1];

		chunkSizesArray[x] = nextBase - currentBase;
	}
}

__global__ void CreateChunkSkeleton(
							// Input Array containing the Ray Chunk Sizes
							int* chunkSizesArray,
							// Input Array containing the Ray Chunk Values
							int* sortedChunkValuesArray,
							// Total number of Ray Chunks
							int chunkTotal,
							// Output Array containing the Ray Chunk Arrays skeleton
							int* skeletonArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= chunkTotal)
		return;

	skeletonArray[x] = chunkSizesArray[sortedChunkValuesArray[x]];
}

__global__ void ClearSortedRays(
							// Total number of Rays
							int rayTotal,
							// Output Arrays containing the sorted Ray Indices
							int* sortedRayIndexKeysArray, 
							int* sortedRayIndexValuesArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= rayTotal)
		return;

	sortedRayIndexKeysArray[x] = 0;
	sortedRayIndexValuesArray[x] = 0;
}

__global__ void CreateSortedRays(
							// Input Arrays containing the Ray Chunk Bases and Sizes
							int* chunkBasesArray,
							int* chunkSizesArray,
							// Input Array containing the chunk hashes and positions
							int* sortedChunkKeysArray,
							int* sortedChunkValuesArray,
							// Input Array containing the inclusive segmented scan result
							int* scanArray, 
							// Total number of Ray Chunks
							int chunkTotal,
							// Auxiliary Array containing the Ray Chunk Arrays head flags 
							int* headFlagsArray, 
							// Auxiliary Array containing the Ray Chunk Arrays skeleton
							int* skeletonArray,
							// Output Arrays containing the sorted Ray Indices
							int* sortedRayIndexKeysArray, 
							int* sortedRayIndexValuesArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= chunkTotal)
		return;

	int chunkKey = sortedChunkKeysArray[x];
	int chunkValue = sortedChunkValuesArray[x];

	int chunkBase = chunkBasesArray[chunkValue];
	int chunkSize = chunkSizesArray[chunkValue];

	int startingPosition = scanArray[x];
	int finalPosition = startingPosition + chunkSize;

	sortedRayIndexKeysArray[startingPosition] = chunkKey;
	sortedRayIndexValuesArray[startingPosition] = chunkBase;

	// Remaining Positions
	for(int i=startingPosition+1, j=1; i<finalPosition; i++) {

		sortedRayIndexKeysArray[i] = chunkKey;
		sortedRayIndexValuesArray[i] = chunkBase + j;
	}
}

__global__ void CreateHierarchyLevel1(	
							// Input Array containing the Rays
							float3* rayArray,
							// Input Arrays containing the trimmed Ray Indices
							int* trimmedRayIndexKeysArray, 
							int* trimmedRayIndexValuesArray,
							// Input Arrays containing the sorted Ray Indices
							int* sortedRayIndexKeysArray, 
							int* sortedRayIndexValuesArray,
							// Total number of Nodes
							int nodeTotal,
							// Output Array containing the Ray Hierarchy
							float4* hierarchyArray) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= nodeTotal)
		return;

	float3 coneDirection = rayArray[trimmedRayIndexValuesArray[sortedRayIndexValuesArray[x]] * 2 + 1];
	float coneSpread = 0.0f;

	float3 sphereCenter = rayArray[trimmedRayIndexValuesArray[sortedRayIndexValuesArray[x]] * 2];
	float sphereRadius = 0.0f;
	
	for(int i=1; i<3; i++) {

		float3 currentConeDirection = rayArray[trimmedRayIndexValuesArray[sortedRayIndexValuesArray[x + i]] * 2 + 1];
		float currentConeSpread = acos(dot(coneDirection, currentConeDirection));
	
		coneDirection = normalize(coneDirection + currentConeDirection);
		coneSpread = currentConeSpread + max(coneSpread, currentConeSpread);
		
		float3 currentSphereCenter = rayArray[trimmedRayIndexValuesArray[sortedRayIndexValuesArray[x + i]] * 2];
		float currentSphereRadius = 0.0f;

		sphereCenter = sphereCenter + normalize(sphereCenter - currentSphereCenter) * length(sphereCenter - currentSphereCenter);
		sphereRadius = length(sphereCenter - currentSphereCenter) * 0.5f + max(sphereRadius, currentSphereRadius);
	}

	hierarchyArray[x] = make_float4(coneDirection.x, coneDirection.y, coneDirection.z, coneSpread);
}

__global__ void CreateHierarchyLevelN(	
							// Input and Output Array containing the Ray Hierarchy
							float4* hierarchyArray,
							// Hierarchy current Level
							int hierarchyLevel,
							// Total number of Nodes
							int nodeTotal) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= nodeTotal)
		return;
}

// Implementation of Whitteds Ray-Tracing Algorithm
__global__ void RayTracePixel(	unsigned int* pixelBufferObject,
								// Screen Dimensions
								const int windowWidth, 
								const int windowHeight,
								// Updated Triangle Position Array
								float4* trianglePositionsArray,
								// Updated Triangle Position Array
								float4* triangleNormalsArray,
								// Input Arrays containing the unsorted Ray Indices
								int* rayIndexKeysArray, 
								int* rayIndexValuesArray,
								// Input Array containing the unsorted Rays
								float3* rayArray,
								// Total Number of Triangles in the Scene
								const int triangleTotal,
								// Total Number of Lights in the Scene
								const int lightTotal,
								// Ray Bounce Depth
								const int depth,
								// Medium Refraction Index
								const float refractionIndex,
								// Camera Definitions
								const float3 cameraPosition) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;		

	if(x >= windowWidth || y >= windowHeight)
		return;

	// Ray Creation
	float3 rayOrigin = make_float3(tex2D(fragmentPositionTexture, x,y));
	float3 rayDirection = reflect(normalize(rayOrigin-cameraPosition), normalize(make_float3(tex2D(fragmentNormalTexture, x,y))));

	if(length(rayOrigin) != 0.0f) {
			
		// Calculate the Final Color
		float3 finalColor = normalize(rayOrigin);
		//float3 finalColor = rayArray[x + y * windowWidth];

		// Update the Pixel Buffer
		pixelBufferObject[y * windowWidth + x] = rgbToInt(finalColor.x * 255, finalColor.y * 255, finalColor.z * 255);
	}
	else {

		// Update the Pixel Buffer
		pixelBufferObject[y * windowWidth + x] = rgbToInt(0.0f, 0.0f, 0.0f);
	}
}

extern "C" {

	void TriangleUpdateWrapper(	// Array containing the updated Model Matrices
								float* modelMatricesArray,
								// Array containing the updated Normal Matrices
								float* normalMatricesArray,
								// Array containing the updated Triangle Positions
								float4* trianglePositionsArray,
								// Array containing the updated Triangle Normals
								float4* triangleNormalsArray,
								// Total Number of Triangles in the Scene
								int triangleTotal) {
		
		// Grid based on the Triangle Count
		dim3 multiplicationBlock(1024);
		dim3 multiplicationGrid(triangleTotal*3/multiplicationBlock.x + 1);
		
		// Model and Normal Matrix Multiplication
		UpdateVertex<<<multiplicationBlock, multiplicationGrid>>>(modelMatricesArray, normalMatricesArray, trianglePositionsArray, triangleNormalsArray, triangleTotal * 3);
	}

	void RayCreationWrapper(
							// Input Array containing the unsorted Rays
							float3* rayArray,
							// Screen Dimensions
							int windowWidth, int windowHeight,
							// Total number of Light Sources in the Scene
							int lightTotal,
							// Cameras Position in the Scene
							float3 cameraPosition,
							// Output Array containing the exclusing scan result
							int* headFlagsArray, 
							// Output Arrays containing the unsorted Ray Indices
							int* rayIndexKeysArray, 
							int* rayIndexValuesArray) {

		// Grid based on the Pixel Count
		dim3 block(32,32);
		dim3 grid(windowWidth/block.x + 1,windowHeight/block.y + 1);

		// Create the Rays
		CreateRays<<<block, grid>>>(rayArray, windowWidth, windowHeight, lightTotal, cameraPosition, headFlagsArray, rayIndexKeysArray, rayIndexValuesArray);
	}

	void RayTrimmingWrapper(	
							// Input Arrays containing the unsorted Ray Indices
							int* rayIndexKeysArray, 
							int* rayIndexValuesArray,
							// Screen Dimensions
							int windowWidth, int windowHeight,
							// Auxiliary Array containing the head flags
							int* headFlagsArray, 
							// Auxiliary Array containing the exclusing scan result
							int* scanArray, 
							// Output Arrays containing the sorted Ray Indices
							int* trimmedRayIndexKeysArray, 
							int* trimmedRayIndexValuesArray) {
	
		// Number of Rays potentialy being cast per Frame
		int rayTotal = windowWidth * windowHeight * RAYS_PER_PIXEL_MAXIMUM;

		// Prepare the Inclusive Scan
		if(scanTemporaryStorage == NULL) {

			// Check how much memory is necessary
			Utility::checkCUDAError("hipcub::DeviceScan::InclusiveSum()", hipcub::DeviceScan::InclusiveSum(scanTemporaryStorage, scanTemporaryStoreBytes, headFlagsArray, scanArray, rayTotal));
			// Allocate temporary storage for exclusive prefix scan
			Utility::checkCUDAError("hipMalloc()", hipMalloc(&scanTemporaryStorage, scanTemporaryStoreBytes));
		}

		// Create the Trim Scan Array
		Utility::checkCUDAError("hipcub::DeviceScan::InclusiveSum()", hipcub::DeviceScan::InclusiveSum(scanTemporaryStorage, scanTemporaryStoreBytes, headFlagsArray, scanArray, rayTotal));

		// Number of Pixels per Frame
		int screenDimensions = windowWidth * windowHeight;

		// Grid based on the Pixel Count
		dim3 block(1024);
		dim3 grid(screenDimensions/block.x + 1);	

		// Trim the Ray Indices Array
		TrimRays<<<block, grid>>>(rayIndexKeysArray, rayIndexValuesArray, screenDimensions, scanArray, trimmedRayIndexKeysArray, trimmedRayIndexValuesArray);
	}

	void RayCompressionWrapper(	
							// Input Arrays containing the trimmed Ray Indices
							int* trimmedRayIndexKeysArray, 
							int* trimmedRayIndexValuesArray,
							// Total number of Rays
							int rayTotal,
							// Auxiliary Array containing the head flags result
							int* headFlagsArray, 
							// Auxiliary Array containing the exclusing scan result
							int* scanArray, 
							// Output Arrays containing the Ray Chunk Bases and Sizes
							int* chunkBasesArray,
							int* chunkSizesArray,
							// Output Arrays containing the Ray Chunks
							int* chunkIndexKeysArray, 
							int* chunkIndexValuesArray) {

		// Grid based on the Ray Count
		dim3 rayBlock(1024);
		dim3 rayGrid(rayTotal/CHUNK_DIVISION/rayBlock.x + 1);

		// Create the Chunk Flags
		CreateChunkFlags<<<rayBlock, rayGrid>>>(trimmedRayIndexKeysArray, trimmedRayIndexValuesArray, rayTotal, headFlagsArray);

		// Prepare the Exclusive Scan
		if(scanTemporaryStorage == NULL) {

			// Check how much memory is necessary
			Utility::checkCUDAError("hipcub::DeviceScan::ExclusiveSum()", hipcub::DeviceScan::InclusiveSum(scanTemporaryStorage, scanTemporaryStoreBytes, headFlagsArray, scanArray, rayTotal));
			// Allocate temporary storage for exclusive prefix scan
			Utility::checkCUDAError("hipMalloc()", hipMalloc(&scanTemporaryStorage, scanTemporaryStoreBytes));
		}

		// Update the Scan Array with each Chunks 
		Utility::checkCUDAError("hipcub::DeviceScan::ExclusiveSum()", hipcub::DeviceScan::InclusiveSum(scanTemporaryStorage, scanTemporaryStoreBytes, headFlagsArray, scanArray, rayTotal));

		int chunkTotal;
		// Check the Ray Total (last position of the scan array)
		Utility::checkCUDAError("hipMemcpy()", hipMemcpy(&chunkTotal, &scanArray[rayTotal-1], sizeof(int), hipMemcpyDeviceToHost));

		// Create the Chunk Bases
		CreateChunkBases<<<rayBlock, rayGrid>>>(trimmedRayIndexKeysArray, trimmedRayIndexValuesArray, rayTotal, headFlagsArray, scanArray, chunkBasesArray, chunkIndexKeysArray, chunkIndexValuesArray);

		// Grid based on the Ray Chunk Count
		dim3 chunkBlock(1024);
		dim3 chunkGrid(chunkTotal/chunkBlock.x + 1);
		
		// Create the Chunk Sizes
		CreateChunkSizes<<<chunkBlock, chunkGrid>>>(chunkBasesArray, chunkTotal, rayTotal, chunkSizesArray);
	}

	void RaySortingWrapper(	
							// Input Arrays containing the Ray Chunks
							int* chunkIndexKeysArray, 
							int* chunkIndexValuesArray,
							// Total number of Ray Chunks
							int chunkTotal,
							// Output Arrays containing the Ray Chunks
							int* sortedChunkIndexKeysArray, 
							int* sortedChunkIndexValuesArray) {

		// Prepare the Radix Sort by allocating temporary storage
		if(radixSortTemporaryStorage == NULL) {

			int total = 768 * 768 * RAYS_PER_PIXEL_MAXIMUM;

			// Check how much memory is necessary
			Utility::checkCUDAError("hipcub::DeviceRadixSort::SortPairs1()", 
				hipcub::DeviceRadixSort::SortPairs(radixSortTemporaryStorage, radixSortTemporaryStoreBytes,
				chunkIndexKeysArray, sortedChunkIndexKeysArray,
				chunkIndexValuesArray, sortedChunkIndexValuesArray, 
				total));
			// Allocate the temporary storage
			Utility::checkCUDAError("hipMalloc()", hipMalloc(&radixSortTemporaryStorage, radixSortTemporaryStoreBytes));
		}
					
		// Run sorting operation
		Utility::checkCUDAError("hipcub::DeviceRadixSort::SortPairs2()", 
			hipcub::DeviceRadixSort::SortPairs(radixSortTemporaryStorage, radixSortTemporaryStoreBytes,
			chunkIndexKeysArray, sortedChunkIndexKeysArray,
			chunkIndexValuesArray, sortedChunkIndexValuesArray, 
			chunkTotal));
	}

	void RayDecompressionWrapper(	
							// Input Arrays containing the Ray Chunk Bases and Sizes
							int* chunkBasesArray,
							int* chunkSizesArray,
							// Input Arrays containing the Ray Chunks
							int* sortedChunkIndexKeysArray, 
							int* sortedChunkIndexValuesArray,
							// Total number of Ray Chunks
							int chunkTotal,
							// Auxiliary Array containing the Ray Chunk Arrays head flags 
							int* headFlagsArray, 
							// Auxiliary Array containing the Ray Chunk Arrays skeleton
							int* skeletonArray,
							// Auxiliary Array containing the inclusive segmented scan result
							int* scanArray, 
							// Output Arrays containing the sorted Ray Indices
							int* sortedRayIndexKeysArray, 
							int* sortedRayIndexValuesArray) {

		// Grid based on the Ray Chunk Count
		dim3 chunkBlock(1024);
		dim3 chunkGrid(chunkTotal/chunkBlock.x + 1);

		CreateChunkSkeleton<<<chunkBlock, chunkGrid>>>(
			chunkSizesArray, 
			sortedChunkIndexValuesArray,
			chunkTotal, 
			skeletonArray);

		// Prepare the Exclusive Scan
		if(scanTemporaryStorage == NULL) {

			// Check how much memory is necessary
			Utility::checkCUDAError("hipcub::DeviceScan::ExclusiveSum()", hipcub::DeviceScan::ExclusiveSum(scanTemporaryStorage, scanTemporaryStoreBytes, skeletonArray, scanArray, chunkTotal));
			// Allocate temporary storage for exclusive prefix scan
			Utility::checkCUDAError("hipMalloc()", hipMalloc(&scanTemporaryStorage, scanTemporaryStoreBytes));
		}

		// Update the Scan Array with each Chunks 
		Utility::checkCUDAError("hipcub::DeviceScan::ExclusiveSum()", hipcub::DeviceScan::ExclusiveSum(scanTemporaryStorage, scanTemporaryStoreBytes, skeletonArray, scanArray, chunkTotal));

		// Create the Chunk Bases
		CreateSortedRays<<<chunkBlock, chunkGrid>>>(
			chunkBasesArray, chunkSizesArray, 
			sortedChunkIndexKeysArray, sortedChunkIndexValuesArray,
			scanArray, 
			chunkTotal, 
			headFlagsArray, 
			skeletonArray, 
			sortedRayIndexKeysArray, sortedRayIndexValuesArray);
	}

	void HierarchyCreationWrapper(	
							// Input Arrays containing the Rays
							float3* rayArray, 
							// Input Arrays containing the trimmed Ray Indices
							int* trimmedRayIndexKeysArray, 
							int* trimmedRayIndexValuesArray,
							// Input Arrays containing the sorted Ray Indices
							int* sortedRayIndexKeysArray, 
							int* sortedRayIndexValuesArray,
							// Total number of Rays
							int rayTotal,
							// Auxiliary Array containing the Ray Chunk Arrays head flags 
							int* headFlagsArray, 
							// Auxiliary Array containing the Ray Chunk Arrays skeleton
							int* skeletonArray,
							// Auxiliary Array containing the inclusive segmented scan result
							int* scanArray, 
							// Output Array containing the Ray Hierarchy
							float4* hierarchyArray) {

		int hierarchyNodeTotal = rayTotal/4 + (rayTotal % 4 != 0 ? 1 : 0);
								
		// Grid based on the Hierarchy Node Count
		dim3 baseLevelBlock(1024);
		dim3 baseLevelGrid(hierarchyNodeTotal/baseLevelBlock.x + 1);

		CreateHierarchyLevel1<<<baseLevelBlock, baseLevelGrid>>>(
			rayArray,
			trimmedRayIndexKeysArray, trimmedRayIndexValuesArray,
			sortedRayIndexKeysArray, sortedRayIndexValuesArray, 
			hierarchyNodeTotal, 
			hierarchyArray);

		cout << "Nodes : " << hierarchyNodeTotal << " Grid: " << baseLevelGrid.x << " Block: " << baseLevelBlock.x << endl;
		
		for(int hierarchyLevel=1; hierarchyLevel<HIERARCHY_MAXIMUM_DEPTH; hierarchyLevel++) {

			hierarchyNodeTotal = hierarchyNodeTotal/4 + (hierarchyNodeTotal % 4 != 0 ? 1 : 0);
			
			// Grid based on the Hierarchy Node Count
			dim3 nLevelBlock(1024);
			dim3 nLevelGrid(hierarchyNodeTotal/baseLevelBlock.x + 1);

			CreateHierarchyLevelN<<<nLevelBlock, nLevelGrid>>>(hierarchyArray, hierarchyLevel, hierarchyNodeTotal);
			
			//cout << "Nodes : " << hierarchyNodeTotal << " Grid: " << nLevelBlock.x << " Block: " << nLevelGrid.x << endl;
		}
	}

	void HierarchyTraversalWrapper(	
							// Input Array containing the Ray Hierarchy
							int* hierarchyArray,
							// Total number of Rays
							int rayTotal,
							// Auxiliary Array containing the Ray Chunk Arrays head flags 
							int* headFlagsArray, 
							// Auxiliary Array containing the Ray Chunk Arrays skeleton
							int* skeletonArray,
							// Auxiliary Array containing the inclusive segmented scan result
							int* scanArray, 
							// Output Array containing the Triangle Hits
							int* hierarchyHitsArray) {
	}

	void IntersectionWrapper(	
							// Input Array containing the Triangle Hits
							int* hierarchyHitsArray,
							// Total number of Rays
							int rayTotal) {
	}

	void RayTraceWrapper(	unsigned int *pixelBufferObject,
							// Screen Dimensions
							int width, int height, 
							// Updated Triangle Position Array
							float4* trianglePositionsArray,
							// Updated Triangle Position Array
							float4* triangleNormalsArray,
							// Input Arrays containing the unsorted Ray Indices
							int* rayIndexKeysArray, 
							int* rayIndexValuesArray,
							// Input Array containing the unsorted Rays
							float3* rayArray,
							// Total Number of Triangles in the Scene
							int triangleTotal,
							// Total Number of Lights in the Scene
							int lightTotal,
							// Camera Definitions
							float3 cameraPosition) {

		// Ray-Casting
		dim3 rayCastingBlock(32,32);
		dim3 rayCastingGrid(width/rayCastingBlock.x + 1,height/rayCastingBlock.y + 1);

		RayTracePixel<<<rayCastingBlock, rayCastingGrid>>>(	pixelBufferObject,
															width, height,
															trianglePositionsArray, 
															triangleNormalsArray,
															rayIndexKeysArray,
															rayIndexValuesArray,
															rayArray,
															triangleTotal,
															lightTotal,
															depth, refractionIndex,
															cameraPosition);

		/*unsigned int bit_mask_1_4 = 15;
		unsigned int bit_mask_1_5 = 31;
		unsigned int bit_mask_1_9 = 511;

		unsigned int half_bit_mask_1_4 = 7;
		unsigned int half_bit_mask_1_5 = 15;
		unsigned int half_bit_mask_1_9 = 255;

		float bit_mask_1_4_f = 15.0f;
		float bit_mask_1_5_f = 31.0f;
		float bit_mask_1_9_f = 511.0f;

		float half_bit_mask_1_4_f = 7.0f;
		float half_bit_mask_1_5_f = 15.0f;
		float half_bit_mask_1_9_f = 255.0f;

		float3 origin = make_float3(5.0f, 5.0f, 5.0f);
		float3 direction = make_float3(0.333f, 0.333f, 0.333f);

		unsigned int index = 0;
			
		int azimuth = (int)clamp((atan(direction.y / direction.x) + HALF_PI) * RADIANS_TO_DEGREES * 2.0f, 0.0f, 360.0f);
		index = azimuth; 
		printf("Azimuth = %u (%u)\n", azimuth, index);

		int polar = (int)clamp(acos(direction.z) * RADIANS_TO_DEGREES, 0.0f, 180.0f);
		index = (index << 9) | polar;
		printf("Polar = %u (%u)\n", polar, index);

		// Clamp the Origin to the 0-15 range
		int x = (int)clamp(origin.x + bit_mask_1_4 / 2, 0.0f, (float)bit_mask_1_4);
		index = (index << 4) | x;
		printf("Coordinate 1 = %u (%u)\n", x, index);
		int y = (int)clamp(origin.y + bit_mask_1_5 / 2, 0.0f, (float)bit_mask_1_5);
		index = (index << 5) | y;
		printf("Coordinate 2 = %u (%u)\n", y, index);
		int z = (int)clamp(origin.z + bit_mask_1_5 / 2, 0.0f, (float)bit_mask_1_5);
		index = (index << 5) | z;
		printf("Coordinate 3 = %u (%u)\n", z, index);
		
		printf("[R] Index = %u\n", index);
		printf("[R] Azimuth = %u (at %u)\n", (index & (bit_mask_1_9 << 23)) >> 23, bit_mask_1_9 << 23);		
		printf("[R] Polar = %u (at %u)\n", (index & (bit_mask_1_9 << 14)) >> 14, bit_mask_1_9 << 14);
		printf("[R] Coordinate 1 = %u (at %u)\n", (index & (bit_mask_1_4 << 10)) >> 10, bit_mask_1_4 << 10);
		printf("[R] Coordinate 2 = %u (at %u)\n", (index & (bit_mask_1_5 << 5)) >> 5, bit_mask_1_5 << 5);
		printf("[R] Coordinate 3 = %u (at %u)\n", (index & bit_mask_1_5), bit_mask_1_5);

		// Convert the Direction to Spherical Coordinates
		index = (int)clamp((atan(direction.y / direction.x) + HALF_PI) * RADIANS_TO_DEGREES * 2.0f, 0.0f, 360.0f);

		index = (index << 9) | (int)clamp(acos(direction.z) * RADIANS_TO_DEGREES, 0.0f, 180.0f);

		// Clamp the Origin to the 0-15 range
		index = (index << 4) | (int)clamp(origin.x + half_bit_mask_1_4_f , 0.0f, bit_mask_1_4_f);
		index = (index << 5) | (int)clamp(origin.y + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);
		index = (index << 5) | (int)clamp(origin.z + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);
		
		printf("[R] Index = %u\n", index);
		printf("[R] Azimuth = %u (at %u)\n", (index & (bit_mask_1_9 << 23)) >> 23, bit_mask_1_9 << 23);		
		printf("[R] Polar = %u (at %u)\n", (index & (bit_mask_1_9 << 14)) >> 14, bit_mask_1_9 << 14);
		printf("[R] Coordinate 1 = %u (at %u)\n", (index & (bit_mask_1_4 << 10)) >> 10, bit_mask_1_4 << 10);
		printf("[R] Coordinate 2 = %u (at %u)\n", (index & (bit_mask_1_5 << 5)) >> 5, bit_mask_1_5 << 5);
		printf("[R] Coordinate 3 = %u (at %u)\n", (index & bit_mask_1_5), bit_mask_1_5);*/
	}

	// OpenGL Texture Binding Functions
	void bindDiffuseTextureArray(hipArray *diffuseTextureArray) {
	
		diffuseTexture.normalized = false;					// access with normalized texture coordinates
		diffuseTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		diffuseTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		diffuseTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(diffuseTexture, diffuseTextureArray, channelDescriptor);
	}

	void bindSpecularTextureArray(hipArray *specularTextureArray) {
	
		specularTexture.normalized = false;					// access with normalized texture coordinates
		specularTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		specularTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		specularTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(specularTexture, specularTextureArray, channelDescriptor);
	}

	void bindFragmentPositionArray(hipArray *fragmentPositionArray) {
	
		fragmentPositionTexture.normalized = false;					// access with normalized texture coordinates
		fragmentPositionTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		fragmentPositionTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		fragmentPositionTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(fragmentPositionTexture, fragmentPositionArray, channelDescriptor);
	}

	void bindFragmentNormalArray(hipArray *fragmentNormalArray) {
	
		fragmentNormalTexture.normalized = false;					// access with normalized texture coordinates
		fragmentNormalTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		fragmentNormalTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		fragmentNormalTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(fragmentNormalTexture, fragmentNormalArray, channelDescriptor);
	}

	// CUDA Triangle Texture Binding Functions
	void bindTrianglePositions(float *cudaDevicePointer, unsigned int triangleTotal) {

		trianglePositionsTexture.normalized = false;                      // access with normalized texture coordinates
		trianglePositionsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		trianglePositionsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, trianglePositionsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleNormals(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleNormalsTexture.normalized = false;                      // access with normalized texture coordinates
		triangleNormalsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleNormalsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, triangleNormalsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleTextureCoordinates(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleTextureCoordinatesTexture.normalized = false;                      // access with normalized texture coordinates
		triangleTextureCoordinatesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleTextureCoordinatesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float2) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float2>();
		hipBindTexture(0, triangleTextureCoordinatesTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleObjectIDs(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleMaterialIDsTexture.normalized = false;                      // access with normalized texture coordinates
		triangleMaterialIDsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleMaterialIDsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(int1) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<int1>();
		hipBindTexture(0, triangleObjectIDsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleMaterialIDs(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleMaterialIDsTexture.normalized = false;                      // access with normalized texture coordinates
		triangleMaterialIDsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleMaterialIDsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(int1) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<int1>();
		hipBindTexture(0, triangleMaterialIDsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	// CUDA Material Texture Binding Functions
	void bindMaterialDiffuseProperties(float *cudaDevicePointer, unsigned int materialTotal) {

		materialDiffusePropertiesTexture.normalized = false;                      // access with normalized texture coordinates
		materialDiffusePropertiesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		materialDiffusePropertiesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * materialTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, materialDiffusePropertiesTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindMaterialSpecularProperties(float *cudaDevicePointer, unsigned int materialTotal) {

		materialSpecularPropertiesTexture.normalized = false;                      // access with normalized texture coordinates
		materialSpecularPropertiesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		materialSpecularPropertiesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * materialTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, materialSpecularPropertiesTexture, cudaDevicePointer, channelDescriptor, size);
	}

	// CUDA Light Texture Binding Functions
	void bindLightPositions(float *cudaDevicePointer, unsigned int lightTotal) {

		lightPositionsTexture.normalized = false;                      // access with normalized texture coordinates
		lightPositionsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		lightPositionsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * lightTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, lightPositionsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindLightColors(float *cudaDevicePointer, unsigned int lightTotal) {

		lightColorsTexture.normalized = false;                      // access with normalized texture coordinates
		lightColorsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		lightColorsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * lightTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, lightColorsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindLightIntensities(float *cudaDevicePointer, unsigned int lightTotal) {

		lightIntensitiesTexture.normalized = false;                      // access with normalized texture coordinates
		lightIntensitiesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		lightIntensitiesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float2) * lightTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float2>();
		hipBindTexture(0, lightIntensitiesTexture, cudaDevicePointer, channelDescriptor, size);
	}
}