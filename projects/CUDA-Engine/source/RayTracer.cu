#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"

#include "math_functions.h"

#include "hip/hip_vector_types.h"

#include "hip/hip_vector_types.h"
#include "vector_functions.h"

// Ray initial depth 
const int initialDepth = 3;
// Ray initial refraction index
const float initialRefractionIndex = 1.0f;

// OpenGL Rendering Texture
texture<uchar4, hipTextureType2D, hipReadModeElementType> renderTexture;

// OpenGL Ray Origin, Reflection and Refraction Textures
texture<float4, hipTextureType2D, hipReadModeElementType> rayOriginTexture;
texture<float4, hipTextureType2D, hipReadModeElementType> rayReflectionTexture;
texture<float4, hipTextureType2D, hipReadModeElementType> rayRefractionTexture;

// Ray testing Constant
__device__ const float epsilon = 0.01f;

// Converts 8-bit integer to floating point rgb color
__device__ float3 intToRgb(int color) {

	float red	= color & 255;
	float green	= (color >> 8) & 255;
	float blue	= (color >> 16) & 255;

	return make_float3(red, green, blue);
}

// Converts floating point rgb color to 8-bit integer
__device__ int rgbToInt(float red, float green, float blue) {

	red		= clamp(red,	0.0f, 255.0f);
	green	= clamp(green,	0.0f, 255.0f);
	blue	= clamp(blue,	0.0f, 255.0f);

	return (int(red)<<16) | (int(green)<<8) | int(blue); // notice switch red and blue to counter the GL_BGRA
}

// Implementation of Whitteds Ray-Tracing Algorithm
__global__ void RayTracePixel(	unsigned int* pixelBufferObject,
								// Screen Dimensions
								const int width, 
								const int height, 
								// Total Number of Triangles in the Scene
								const int triangleTotal,
								// Total Number of Lights in the Scene
								const int lightTotal,
								// Ray Bounce Depth
								const int depth,
								// Medium Refraction Index
								const float refractionIndex,
								// Camera Definitions
								const float3 cameraPosition, 
								const float3 cameraUp, const float3 cameraRight, const float3 cameraDirection) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int3 pixelColor = make_int3(0);

	pixelColor.x = 255 - tex2D(renderTexture, x, y).x;
	pixelColor.y = 255 - tex2D(renderTexture, x, y).y;
	pixelColor.z = 255 - tex2D(renderTexture, x, y).z;

	int rgb = pixelColor.x;
	rgb = (rgb << 8) + pixelColor.y;
	rgb = (rgb << 8) + pixelColor.z;

	pixelBufferObject[y * width + x] = rgb;

	float4 color = tex2D(rayReflectionTexture, x, y);
	pixelBufferObject[y * width + x] = rgbToInt((color.x + 1.0f)* 128.0f, (color.y + 1.0f) * 128.0f, (color.z + 1.0f) * 128.0f);
}

extern "C" {

	void RayTraceWrapper(	unsigned int *pixelBufferObject,
								int width, int height, 
								int triangleTotal,
								int lightTotal,
								float3 cameraPosition,
								float3 cameraUp, float3 cameraRight, float3 cameraDirection
								) {

		dim3 block(8,8,1);
		dim3 grid(width/block.x,height/block.y, 1);

		RayTracePixel<<<grid, block>>>(	pixelBufferObject,
										width, height,
										triangleTotal,
										lightTotal,
										initialDepth,
										initialRefractionIndex,
										cameraPosition,
										cameraUp, cameraRight, cameraDirection);
	}

	// OpenGL Texture Binding Functions
	void bindRenderTextureArray(hipArray *renderArray) {
	
		renderTexture.normalized = false;					// access with normalized texture coordinates
		renderTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		renderTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		renderTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<uchar4>();
		hipBindTextureToArray(renderTexture, renderArray, channelDescriptor);
	}

	// OpenGL Texture Binding Functions
	void bindRayOriginTextureArray(hipArray *rayOriginArray) {
	
		rayOriginTexture.normalized = false;					// access with normalized texture coordinates
		rayOriginTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		rayOriginTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		rayOriginTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(rayOriginTexture, rayOriginArray, channelDescriptor);
	}

	void bindRayReflectionTextureArray(hipArray *rayReflectionArray) {
	
		rayReflectionTexture.normalized = false;					// access with normalized texture coordinates
		rayReflectionTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		rayReflectionTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		rayReflectionTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(rayReflectionTexture, rayReflectionArray, channelDescriptor);
	}

	void bindRayRefractionTextureArray(hipArray *rayRefractionArray) {
	
		rayRefractionTexture.normalized = false;					// access with normalized texture coordinates
		rayRefractionTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		rayRefractionTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		rayRefractionTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(rayRefractionTexture, rayRefractionArray, channelDescriptor);
	}
}