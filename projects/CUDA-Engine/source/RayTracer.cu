#include "hip/hip_runtime.h"
// CUDA definitions
#include <hip/hip_runtime.h>
// CUB definitions
#include <cub.cuh>

// Math Includes 
#include <hip/hip_vector_types.h>
#include <math_functions.h>
// Vector Includes
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

// C++ Includes
#include <stdio.h>
// Utility Includes
#include "Utility.h"
#include "Constants.h"

// Secondary Ray Depth
static const int depth = 0;
// Air Refraction Index
static const float refractionIndex = 1.0f;

// Ray testing Constant
static const float epsilon = 0.01f;

// Light Maximum Amount
static const int lightSourceMaximum = 10;
static const int raysPerPixel = 12;

// Ray indexing Constants
__constant__ __device__ static const unsigned int bit_mask_1_4 = 15;
__constant__ __device__ static const unsigned int bit_mask_1_5 = 31;
__constant__ __device__ static const unsigned int bit_mask_1_9 = 511;

__constant__ __device__ static const unsigned int half_bit_mask_1_4 = 7;
__constant__ __device__ static const unsigned int half_bit_mask_1_5 = 15;
__constant__ __device__ static const unsigned int half_bit_mask_1_9 = 255;

__constant__ __device__ static const float bit_mask_1_4_f = 15.0f;
__constant__ __device__ static const float bit_mask_1_5_f = 31.0f;
__constant__ __device__ static const float bit_mask_1_9_f = 511.0f;

__constant__ __device__ static const float half_bit_mask_1_4_f = 7.0f;
__constant__ __device__ static const float half_bit_mask_1_5_f = 15.0f;
__constant__ __device__ static const float half_bit_mask_1_9_f = 255.0f;

// OpenGL Diffuse and Specular Textures
texture<float4, hipTextureType2D, hipReadModeElementType> diffuseTexture;
texture<float4, hipTextureType2D, hipReadModeElementType> specularTexture;
// OpenGL Fragment Position and Normal Textures
texture<float4, hipTextureType2D, hipReadModeElementType> fragmentPositionTexture;
texture<float4, hipTextureType2D, hipReadModeElementType> fragmentNormalTexture;

// CUDA Triangle Textures
texture<float4, 1, hipReadModeElementType> trianglePositionsTexture;
texture<float4, 1, hipReadModeElementType> triangleNormalsTexture;
texture<float2, 1, hipReadModeElementType> triangleTextureCoordinatesTexture;

// CUDA Triangle ID Textures
texture<int1, 1, hipReadModeElementType> triangleObjectIDsTexture;
texture<int1, 1, hipReadModeElementType> triangleMaterialIDsTexture;

// CUDA Material Textures
texture<float4, 1, hipReadModeElementType> materialDiffusePropertiesTexture;
texture<float4, 1, hipReadModeElementType> materialSpecularPropertiesTexture;

// CUDA Light Textures
texture<float4, 1, hipReadModeElementType> lightPositionsTexture;
texture<float4, 1, hipReadModeElementType> lightColorsTexture;
texture<float2, 1, hipReadModeElementType> lightIntensitiesTexture;

// Ray structure
struct Ray {

	float3 origin;
	float3 direction;
	float3 inverseDirection;

	__device__ Ray() {};
	__device__ Ray(const float3 &o,const float3 &d) {

		origin = o;
		direction = d;
		direction = normalize(direction);
		inverseDirection = make_float3(1.0/direction.x, 1.0/direction.y, 1.0/direction.z);
	}
};

struct HitRecord {

	float time;

	float3 color;

	float3 point;
	float3 normal;

	int triangleIndex;

	__device__ HitRecord(const float3 &c) {

			time = UINT_MAX;

			color = c;

			point = make_float3(0,0,0);
			normal = make_float3(0,0,0);

			triangleIndex = -1; 
	}

	__device__ void resetTime() {
		
			time = UINT_MAX;

			point = make_float3(0,0,0);
			normal = make_float3(0,0,0);

			triangleIndex = -1;
	}
};

// Converts 8-bit integer to floating point rgb color
__device__ float3 intToRgb(int color) {

	float red	= color & 255;
	float green	= (color >> 8) & 255;
	float blue	= (color >> 16) & 255;

	return make_float3(red, green, blue);
}

// Converts floating point rgb color to 8-bit integer
__device__ int rgbToInt(float red, float green, float blue) {

	red		= clamp(red,	0.0f, 255.0f);
	green	= clamp(green,	0.0f, 255.0f);
	blue	= clamp(blue,	0.0f, 255.0f);

	return (int(red)) | (int(green)<<8) | (int(blue)<<16); // notice switch red and blue to counter the GL_BGRA
}

// Converts a Direction Vector to Spherical Coordinates
__device__ float2 vectorToSpherical(float3 direction) {

	float azimuth = atan(direction.y / direction.x) * 2.0f;
	float polar = acos(direction.z);

	return make_float2(azimuth,polar);
}

// Converts Spherical Coordinates to a Direction Vector
__device__ float3 sphericalToVector(float2 spherical) {

	float x = cos(spherical.x) * sin(spherical.y);
	float y = sin(spherical.x) * sin(spherical.y);
	float z = cos(spherical.y);

	return make_float3(x,y,z);
}

// Converts a ray to an integer hash value
__device__ int rayToIndex(float3 origin, float3 direction) {

	// 32 bits
	//	- 14 bits for the origin (4 for x, 5 for y and 5 for z)
	//  - 18 bits for the direction (10 for longitude, 10 for latitude)
	int index = 0;

	// Convert the Direction to Spherical Coordinates
	index = (unsigned int)clamp((atan(direction.y / direction.x) + HALF_PI) * RADIANS_TO_DEGREES * 2.0f, 0.0f, 360.0f);
	index = (index << 9) | (unsigned int)clamp(acos(direction.z) * RADIANS_TO_DEGREES, 0.0f, 180.0f);

	// Clamp the Origin to the 0-15 range
	index = (index << 4) | (unsigned int)clamp(origin.x + half_bit_mask_1_4_f , 0.0f, bit_mask_1_4_f);
	index = (index << 5) | (unsigned int)clamp(origin.y + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);
	index = (index << 5) | (unsigned int)clamp(origin.z + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);

	return index;
}


// Ray - BoundingBox Intersection Code
__device__ int RayBoxIntersection(const float3 &BBMin, const float3 &BBMax, const float3 &RayOrigin, const float3 &RayDirectionInverse, float &tmin, float &tmax) {

	float l1   = (BBMin.x - RayOrigin.x) * RayDirectionInverse.x;
	float l2   = (BBMax.x - RayOrigin.x) * RayDirectionInverse.x;
	tmin = fminf(l1,l2);
	tmax = fmaxf(l1,l2);

	l1   = (BBMin.y - RayOrigin.y) * RayDirectionInverse.y;
	l2   = (BBMax.y - RayOrigin.y) * RayDirectionInverse.y;
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);

	l1   = (BBMin.z - RayOrigin.z) * RayDirectionInverse.z;
	l2   = (BBMax.z - RayOrigin.z) * RayDirectionInverse.z;
	tmin = fmaxf(fminf(l1,l2), tmin);
	tmax = fminf(fmaxf(l1,l2), tmax);

	return ((tmax >= tmin) && (tmax >= 0.0f));
}

// Ray - Triangle Intersection Code
__device__ float RayTriangleIntersection(const Ray &ray, const float3 &vertex0, const float3 &edge1, const float3 &edge2) {  

	float3 tvec = ray.origin - vertex0;  
	float3 pvec = cross(ray.direction, edge2);  

	float  determinant  = dot(edge1, pvec);  
	determinant = __fdividef(1.0f, determinant);  

	// First Test
	float u = dot(tvec, pvec) * determinant;  
	if (u < 0.0f || u > 1.0f)  
		return -1.0f;  

	// Second Test
	float3 qvec = cross(tvec, edge1);  

	float v = dot(ray.direction, qvec) * determinant;  
	if (v < 0.0f || (u + v) > 1.0f)  
		return -1.0f;  

	return dot(edge2, qvec) * determinant;  
}  

// Implementation of the Matrix Multiplication
__global__ void MultiplyVertex(
							// Updated Normal Matrices Array
							float* modelMatricesArray,
							// Updated Normal Matrices Array
							float* normalMatricesArray,
							// Updated Triangle Positions Array
							float4* trianglePositionsArray,
							// Updated Triangle Normals Array
							float4* triangleNormalsArray,
							// Total Number of Vertices in the Scene
							int vertexTotal) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	if(x >= vertexTotal)
		return;

	// Matrices ID
	int matrixID = tex1Dfetch(triangleObjectIDsTexture, x).x;

	// Vertices
	float modelMatrix[16];

	for(int i=0; i<16; i++)
		modelMatrix[i] = modelMatricesArray[matrixID * 16 + i];
	
	float4 vertex = tex1Dfetch(trianglePositionsTexture, x);

	float updatedVertex[4];

	for(int i=0; i<4; i++) {

		updatedVertex[i] = 0.0f;
		updatedVertex[i] += modelMatrix[i * 4 + 0] * vertex.x;
		updatedVertex[i] += modelMatrix[i * 4 + 1] * vertex.y;
		updatedVertex[i] += modelMatrix[i * 4 + 2] * vertex.z;
		updatedVertex[i] += modelMatrix[i * 4 + 3] * vertex.w;
	}
	
	trianglePositionsArray[x] = make_float4(updatedVertex[0], updatedVertex[1], updatedVertex[2], matrixID);

	// Normals
	float normalMatrix[16];

	for(int i=0; i<16; i++)
		normalMatrix[i] = normalMatricesArray[matrixID * 16 + i];

	float4 normal = tex1Dfetch(triangleNormalsTexture, x);

	float updatedNormal[4];

	for(int i=0; i<4; i++) {

		updatedNormal[i] = 0.0f;
		updatedNormal[i] += normalMatrix[i * 4 + 0] * normal.x;
		updatedNormal[i] += normalMatrix[i * 4 + 1] * normal.y;
		updatedNormal[i] += normalMatrix[i * 4 + 2] * normal.z;
		updatedNormal[i] += normalMatrix[i * 4 + 3] * normal.w;
	}

	triangleNormalsArray[x] = make_float4(normalize(make_float3(updatedNormal[0], updatedNormal[1], updatedNormal[2])), 0.0f);
}

// Implementation of the Ray Creation and Indexing
__global__ void RayCreation(// Input Array containing the unsorted Rays
							float3* rayArray,
							// Screen Dimensions
							int windowWidth, int windowHeight,
							// Total number of Light Sources in the Scene
							int lightTotal,
							// Cameras Position in the Scene
							float3 cameraPosition,
							// Output Array containing the unsorted Ray Indices
							int2* rayIndicesArray) {

	// Ray Indexing		
	//
	// Use Directional Indexing first (24 bits)
	// Use Positional Indexing second (8 bits)
	//
	// Output
	//		Ray index Array	

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= windowWidth || y >= windowHeight)
		return;

	int rayArrayBase = x * raysPerPixel + y * windowWidth * raysPerPixel;

	// Fragment Position and Normal - Sent from the OpenGL Rasterizer
	float3 fragmentPosition = make_float3(tex2D(fragmentPositionTexture, x,y));
	float3 fragmentNormal = normalize(make_float3(tex2D(fragmentNormalTexture, x,y)));

	// Ray Origin Creation
	float3 rayOrigin = fragmentPosition;

	if(length(rayOrigin) != 0.0f) {
		
		// Ray Direction Creation
		float3 rayReflectionDirection = reflect(normalize(fragmentPosition-cameraPosition), normalize(fragmentNormal));
		float3 rayRefractionDirection = refract(normalize(fragmentPosition-cameraPosition), normalize(fragmentNormal), 1.0f / 1.52f);

		// Create the Reflection Ray and store its direction
		rayArray[rayArrayBase] = rayReflectionDirection;
		rayIndicesArray[rayArrayBase] = make_int2(rayToIndex(rayOrigin, rayReflectionDirection), rayArrayBase);
		rayArrayBase++;

		// Create the Refraction Ray and store its direction
		rayArray[rayArrayBase] = rayRefractionDirection;
		rayIndicesArray[rayArrayBase] = make_int2(rayToIndex(rayOrigin, rayRefractionDirection), rayArrayBase);
		rayArrayBase++;

		for(int l = 0; l < lightSourceMaximum; l++) {

			if(l < lightTotal) {

				// Fetch the Light Position
				float3 lightPosition = make_float3(tex1Dfetch(lightPositionsTexture, l));
				// Calculate the Light Direction
				float3 lightDirection = normalize(lightPosition - fragmentPosition);

				// Diffuse Factor
				float diffuseFactor = max(dot(lightDirection, fragmentNormal), 0.0f);
				clamp(diffuseFactor, 0.0f, 1.0f);
				
				if(diffuseFactor >
					0.0f) {
					
					// Create the Shadow Ray and store its direction
					rayArray[rayArrayBase] = lightDirection;
					rayIndicesArray[rayArrayBase] = make_int2(rayToIndex(lightPosition, lightDirection), rayArrayBase);
					rayArrayBase++;
				}
			}
			else {
				
				// Clean the Shadow Ray storage
				rayArray[rayArrayBase] = make_float3(0.0f);
				rayIndicesArray[rayArrayBase] = make_int2(0);
				rayArrayBase++;
			}
		}
	}
	else {
	
		// Clean the Reflection Ray storage
		rayArray[rayArrayBase] = make_float3(0.0f);
		rayIndicesArray[rayArrayBase] = make_int2(0);
		rayArrayBase++;

		// Clean the Refraction Ray storage
		rayArray[rayArrayBase] = make_float3(0.0f);
		rayIndicesArray[rayArrayBase] = make_int2(0);
		rayArrayBase++;

		// Clean the Shadow Ray storage
		rayArray[rayArrayBase] = make_float3(0.0f);
		rayIndicesArray[rayArrayBase] = make_int2(0);
		rayArrayBase++;
	}
}

// Implementation of the Ray Compression
__global__ void RayCompression(	
							// Input Array containing the unsorted Ray Indices
							int2* rayIndicesArray,
							// Auxiliary Array containing the head flags result
							int* headFlagsArray, 
							// Auxiliary Array containing the exclusing scan result
							int* scanArray, 
							// Output Array containing the unsorted Ray Chunks
							int2* chunkArray) {

	// Ray Compression - Compress Rays with the same index into chunks 
	//
	// Create the Head Flags Array (Initialized with 0)
	//		Head: (ray[i] != ray[i-1] => head[i] = 1 : head[i] = 0)
	//
	// Exclusive Scan on the Head Array (Initialized with 0)
	//		Scan: Sum of the Head Array
	//
	// Create the Chucks and Size Array (Initialized with 0 and 0)
	//		Base: (head[i] != head[i+1] => base[i] = i) 
	//		Size: (size[i] = base[i+1] - base[i])
	// 
	// Output 
	//		Base Array with the starting index of the chunk 
	//		Size Array with the size of the chunk
}

// Implementation of the Ray Sorting
__global__ void RaySorting(	
							// Input Array containing the unsorted Ray Chunks
							int2* chunkArray, 
							// Output Array containing the sorted Ray Chunks
							int2* sortedChunkArray) {

	// Ray Sorting - Radix Sort the Base Array and the Size Array
	//
	// Radix Sort on the Base Array
	//
	// Size Array doesn't have to be sorted, just needs to follow the sorting of the Base Array
}

// Implementation of the Ray Decompression
__global__ void RayDecompression(
							// Input Array containing the sorted Ray Chunks
							int2* sortedChunkArray, 
							// Auxiliary Array containing the Ray Chunk Arrays head flags 
							int* headFlagsArray, 
							// Auxiliary Array containing the Ray Chunk Arrays skeleton
							int* skeletonArray,
							// Auxiliary Array containing the inclusive segmented scan result
							int* scanArray, 
							// Output Array containing the sorted Ray Indices
							int2* sortedRayIndicesArray) {

	// Ray Decompression - Decompress Rays from the sorted chunks
	//
	// Exclusive Scan on the sorted Size Array
	//		Scan: Sum of the sorted Size Array
	//
	// Create the Skeleton and Head Flags Array (Initialized with 1 and 0)
	//		Skeleton: skeleton[i] = base[scan[i]]
	//		Head: head[i] = (skeleton[i] != 0)
	//
	// Inclusive Segmented Scan on the Skeleton and Head Arrays
	//
	// Output 
	//		Sorted ray index Array	
}

// Implementation of Whitteds Ray-Tracing Algorithm
__global__ void RayTracePixel(	unsigned int* pixelBufferObject,
								// Screen Dimensions
								const int width, 
								const int height,
								// Updated Triangle Position Array
								float4* trianglePositionsArray,
								// Updated Triangle Position Array
								float4* triangleNormalsArray,
								// Input Array containing the unsorted Rays
								float3* rayArray,
								// Total Number of Triangles in the Scene
								const int triangleTotal,
								// Total Number of Lights in the Scene
								const int lightTotal,
								// Ray Bounce Depth
								const int depth,
								// Medium Refraction Index
								const float refractionIndex,
								// Camera Definitions
								const float3 cameraPosition) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;		

	if(x >= width || y >= height)
		return;

	// Ray Creation
	float3 rayOrigin = make_float3(tex2D(fragmentPositionTexture, x,y));
	float3 rayDirection = reflect(normalize(rayOrigin-cameraPosition), normalize(make_float3(tex2D(fragmentNormalTexture, x,y))));

	if(length(rayOrigin) != 0.0f) {
			
		// Calculate the Final Color
		float3 finalColor = normalize(rayOrigin);
		//float3 finalColor = rayArray[x * raysPerPixel + y * width * raysPerPixel + 2];

		// Update the Pixel Buffer
		pixelBufferObject[y * width + x] = rgbToInt(finalColor.x * 255, finalColor.y * 255, finalColor.z * 255);
	}
	else {
	
		// Update the Pixel Buffer
		pixelBufferObject[y * width + x] = rgbToInt(0.0f, 0.0f, 0.0f);
	}
}

extern "C" {

	void TriangleUpdateWrapper(	// Array containing the updated Model Matrices
								float* modelMatricesArray,
								// Array containing the updated Normal Matrices
								float* normalMatricesArray,
								// Array containing the updated Triangle Positions
								float4* trianglePositionsArray,
								// Array containing the updated Triangle Normals
								float4* triangleNormalsArray,
								// Total Number of Triangles in the Scene
								int triangleTotal) {
		
		// Grid based on the Triangle Count
		dim3 multiplicationBlock(1024);
		dim3 multiplicationGrid(triangleTotal*3/1024 + 1);
		
		// Model and Normal Matrix Multiplication
		MultiplyVertex<<<multiplicationBlock, multiplicationGrid>>>(modelMatricesArray, normalMatricesArray, trianglePositionsArray, triangleNormalsArray, triangleTotal * 3);
	}

	void RayCreationWrapper(// Input Array containing the unsorted Rays
							float3* rayArray,
							// Screen Dimensions
							int windowWidth, int windowHeight,
							// Total number of Light Sources in the Scene
							int lightTotal,
							// Cameras Position in the Scene
							float3 cameraPosition,
							// Output Array containing the unsorted Ray Indices
							int2* rayIndicesArray) {

		// Grid based on the Pixel Count
		dim3 block(32,32);
		dim3 grid(windowWidth/block.x + 1,windowHeight/block.y + 1);

		RayCreation<<<block, grid>>>(rayArray, windowWidth, windowHeight, lightTotal, cameraPosition, rayIndicesArray);
	}

	void RayCompressionWrapper(	// Input Array containing the unsorted Ray Indices
								int2* rayIndicesArray,
								// Auxiliary Array containing the head flags result
								int* headFlagsArray, 
								// Auxiliary Array containing the exclusing scan result
								int* scanArray, 
								// Output Array containing the unsorted Ray Chunks
								int2* chunkArray) {

		int rayTotal = 768*768*7;
		// Use Exclusing and Inclusive Sums to calculate Array Size and also to Truncate the Arrays

		// Grid based on the Ray Count
		dim3 block(1024);
		dim3 grid(rayTotal/1024 + 1);
		
		// Ray Compression
		RayCompression<<<block, grid>>>(rayIndicesArray, headFlagsArray, scanArray, chunkArray);
	}

	void RaySortingWrapper(	// Input Array containing the unsorted Ray Chunks
							int2* chunkArray, 
							// Output Array containing the sorted Ray Chunks
							int2* sortedChunkArray) {

	}

	void RayDecompressionWrapper(	// Input Array containing the sorted Ray Chunks
									int2* sortedChunkArray, 
									// Auxiliary Array containing the Ray Chunk Arrays head flags 
									int* headFlagsArray, 
									// Auxiliary Array containing the Ray Chunk Arrays skeleton
									int* skeletonArray,
									// Auxiliary Array containing the inclusive segmented scan result
									int* scanArray, 
									// Output Array containing the sorted Ray Indices
									int2* sortedRayIndicesArray) {

		int chunkTotal = 768*768*7;

		// Grid based on the Chunk Count
		dim3 block(1024);
		dim3 grid(chunkTotal/1024 + 1);
		
		// Ray Decompression
		RayDecompression<<<block, grid>>>(sortedChunkArray, headFlagsArray, skeletonArray, scanArray, sortedRayIndicesArray);
	}

	void RayTraceWrapper(	unsigned int *pixelBufferObject,
							// Screen Dimensions
							int width, int height, 			
							// Updated Normal Matrices Array
							float* modelMatricesArray,
							// Updated Normal Matrices Array
							float* normalMatricesArray,
							// Updated Triangle Position Array
							float4* trianglePositionsArray,
							// Updated Triangle Position Array
							float4* triangleNormalsArray,
							// Input Array containing the unsorted Rays
							float3* rayArray,
							// Total Number of Triangles in the Scene
							int triangleTotal,
							// Total Number of Lights in the Scene
							int lightTotal,
							// Camera Definitions
							float3 cameraPosition) {

		// Ray-Casting
		dim3 rayCastingBlock(32,32);
		dim3 rayCastingGrid(width/rayCastingBlock.x + 1,height/rayCastingBlock.y + 1);

		RayTracePixel<<<rayCastingBlock, rayCastingGrid>>>(	pixelBufferObject,
															width, height,
															trianglePositionsArray, 
															triangleNormalsArray,
															rayArray,
															triangleTotal,
															lightTotal,
															depth, refractionIndex,
															cameraPosition);

		/*unsigned int bit_mask_1_4 = 15;
		unsigned int bit_mask_1_5 = 31;
		unsigned int bit_mask_1_9 = 511;

		unsigned int half_bit_mask_1_4 = 7;
		unsigned int half_bit_mask_1_5 = 15;
		unsigned int half_bit_mask_1_9 = 255;

		float bit_mask_1_4_f = 15.0f;
		float bit_mask_1_5_f = 31.0f;
		float bit_mask_1_9_f = 511.0f;

		float half_bit_mask_1_4_f = 7.0f;
		float half_bit_mask_1_5_f = 15.0f;
		float half_bit_mask_1_9_f = 255.0f;

		float3 origin = make_float3(5.0f, 5.0f, 5.0f);
		float3 direction = make_float3(0.333f, 0.333f, 0.333f);

		unsigned int index = 0;
			
		int azimuth = (int)clamp((atan(direction.y / direction.x) + HALF_PI) * RADIANS_TO_DEGREES * 2.0f, 0.0f, 360.0f);
		index = azimuth; 
		printf("Azimuth = %u (%u)\n", azimuth, index);

		int polar = (int)clamp(acos(direction.z) * RADIANS_TO_DEGREES, 0.0f, 180.0f);
		index = (index << 9) | polar;
		printf("Polar = %u (%u)\n", polar, index);

		// Clamp the Origin to the 0-15 range
		int x = (int)clamp(origin.x + bit_mask_1_4 / 2, 0.0f, (float)bit_mask_1_4);
		index = (index << 4) | x;
		printf("Coordinate 1 = %u (%u)\n", x, index);
		int y = (int)clamp(origin.y + bit_mask_1_5 / 2, 0.0f, (float)bit_mask_1_5);
		index = (index << 5) | y;
		printf("Coordinate 2 = %u (%u)\n", y, index);
		int z = (int)clamp(origin.z + bit_mask_1_5 / 2, 0.0f, (float)bit_mask_1_5);
		index = (index << 5) | z;
		printf("Coordinate 3 = %u (%u)\n", z, index);
		
		printf("[R] Index = %u\n", index);
		printf("[R] Azimuth = %u (at %u)\n", (index & (bit_mask_1_9 << 23)) >> 23, bit_mask_1_9 << 23);		
		printf("[R] Polar = %u (at %u)\n", (index & (bit_mask_1_9 << 14)) >> 14, bit_mask_1_9 << 14);
		printf("[R] Coordinate 1 = %u (at %u)\n", (index & (bit_mask_1_4 << 10)) >> 10, bit_mask_1_4 << 10);
		printf("[R] Coordinate 2 = %u (at %u)\n", (index & (bit_mask_1_5 << 5)) >> 5, bit_mask_1_5 << 5);
		printf("[R] Coordinate 3 = %u (at %u)\n", (index & bit_mask_1_5), bit_mask_1_5);

		// Convert the Direction to Spherical Coordinates
		index = (int)clamp((atan(direction.y / direction.x) + HALF_PI) * RADIANS_TO_DEGREES * 2.0f, 0.0f, 360.0f);

		index = (index << 9) | (int)clamp(acos(direction.z) * RADIANS_TO_DEGREES, 0.0f, 180.0f);

		// Clamp the Origin to the 0-15 range
		index = (index << 4) | (int)clamp(origin.x + half_bit_mask_1_4_f , 0.0f, bit_mask_1_4_f);
		index = (index << 5) | (int)clamp(origin.y + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);
		index = (index << 5) | (int)clamp(origin.z + half_bit_mask_1_5_f, 0.0f, bit_mask_1_5_f);
		
		printf("[R] Index = %u\n", index);
		printf("[R] Azimuth = %u (at %u)\n", (index & (bit_mask_1_9 << 23)) >> 23, bit_mask_1_9 << 23);		
		printf("[R] Polar = %u (at %u)\n", (index & (bit_mask_1_9 << 14)) >> 14, bit_mask_1_9 << 14);
		printf("[R] Coordinate 1 = %u (at %u)\n", (index & (bit_mask_1_4 << 10)) >> 10, bit_mask_1_4 << 10);
		printf("[R] Coordinate 2 = %u (at %u)\n", (index & (bit_mask_1_5 << 5)) >> 5, bit_mask_1_5 << 5);
		printf("[R] Coordinate 3 = %u (at %u)\n", (index & bit_mask_1_5), bit_mask_1_5);*/
	}

	// OpenGL Texture Binding Functions
	void bindDiffuseTextureArray(hipArray *diffuseTextureArray) {
	
		diffuseTexture.normalized = false;					// access with normalized texture coordinates
		diffuseTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		diffuseTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		diffuseTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(diffuseTexture, diffuseTextureArray, channelDescriptor);
	}

	void bindSpecularTextureArray(hipArray *specularTextureArray) {
	
		specularTexture.normalized = false;					// access with normalized texture coordinates
		specularTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		specularTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		specularTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(specularTexture, specularTextureArray, channelDescriptor);
	}

	void bindFragmentPositionArray(hipArray *fragmentPositionArray) {
	
		fragmentPositionTexture.normalized = false;					// access with normalized texture coordinates
		fragmentPositionTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		fragmentPositionTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		fragmentPositionTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(fragmentPositionTexture, fragmentPositionArray, channelDescriptor);
	}

	void bindFragmentNormalArray(hipArray *fragmentNormalArray) {
	
		fragmentNormalTexture.normalized = false;					// access with normalized texture coordinates
		fragmentNormalTexture.filterMode = hipFilterModePoint;		// Point mode, so no 
		fragmentNormalTexture.addressMode[0] = hipAddressModeWrap;	// wrap texture coordinates
		fragmentNormalTexture.addressMode[1] = hipAddressModeWrap;	// wrap texture coordinates

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTextureToArray(fragmentNormalTexture, fragmentNormalArray, channelDescriptor);
	}

	// CUDA Triangle Texture Binding Functions
	void bindTrianglePositions(float *cudaDevicePointer, unsigned int triangleTotal) {

		trianglePositionsTexture.normalized = false;                      // access with normalized texture coordinates
		trianglePositionsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		trianglePositionsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, trianglePositionsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleNormals(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleNormalsTexture.normalized = false;                      // access with normalized texture coordinates
		triangleNormalsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleNormalsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, triangleNormalsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleTextureCoordinates(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleTextureCoordinatesTexture.normalized = false;                      // access with normalized texture coordinates
		triangleTextureCoordinatesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleTextureCoordinatesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float2) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float2>();
		hipBindTexture(0, triangleTextureCoordinatesTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleObjectIDs(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleMaterialIDsTexture.normalized = false;                      // access with normalized texture coordinates
		triangleMaterialIDsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleMaterialIDsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(int1) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<int1>();
		hipBindTexture(0, triangleObjectIDsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindTriangleMaterialIDs(float *cudaDevicePointer, unsigned int triangleTotal) {

		triangleMaterialIDsTexture.normalized = false;                      // access with normalized texture coordinates
		triangleMaterialIDsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		triangleMaterialIDsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(int1) * triangleTotal * 3;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<int1>();
		hipBindTexture(0, triangleMaterialIDsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	// CUDA Material Texture Binding Functions
	void bindMaterialDiffuseProperties(float *cudaDevicePointer, unsigned int materialTotal) {

		materialDiffusePropertiesTexture.normalized = false;                      // access with normalized texture coordinates
		materialDiffusePropertiesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		materialDiffusePropertiesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * materialTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, materialDiffusePropertiesTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindMaterialSpecularProperties(float *cudaDevicePointer, unsigned int materialTotal) {

		materialSpecularPropertiesTexture.normalized = false;                      // access with normalized texture coordinates
		materialSpecularPropertiesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		materialSpecularPropertiesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * materialTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, materialSpecularPropertiesTexture, cudaDevicePointer, channelDescriptor, size);
	}

	// CUDA Light Texture Binding Functions
	void bindLightPositions(float *cudaDevicePointer, unsigned int lightTotal) {

		lightPositionsTexture.normalized = false;                      // access with normalized texture coordinates
		lightPositionsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		lightPositionsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * lightTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, lightPositionsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindLightColors(float *cudaDevicePointer, unsigned int lightTotal) {

		lightColorsTexture.normalized = false;                      // access with normalized texture coordinates
		lightColorsTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		lightColorsTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float4) * lightTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float4>();
		hipBindTexture(0, lightColorsTexture, cudaDevicePointer, channelDescriptor, size);
	}

	void bindLightIntensities(float *cudaDevicePointer, unsigned int lightTotal) {

		lightIntensitiesTexture.normalized = false;                      // access with normalized texture coordinates
		lightIntensitiesTexture.filterMode = hipFilterModePoint;        // Point mode, so no 
		lightIntensitiesTexture.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

		size_t size = sizeof(float2) * lightTotal;

		hipChannelFormatDesc channelDescriptor = hipCreateChannelDesc<float2>();
		hipBindTexture(0, lightIntensitiesTexture, cudaDevicePointer, channelDescriptor, size);
	}
}